#include "hip/hip_runtime.h"
#include "blur.h"

__global__
void apply_filter(const uchar *input, uchar *output, const uint rows, const uint cols, const double *kernel,
        const uint kernel_size) {
    uint x = threadIdx.x + blockIdx.x * blockDim.x;
    uint y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x < cols && y < rows) {
        int half = static_cast<int>(kernel_size / 2);
        double blur = 0.0f;

        for (int i = -half; i <= half; i++)
            for (int j = -half; j <= half; j++) {
                uint orig_x = max(0, min(cols - 1, x + j));
                uint orig_y = max(0, min(rows - 1, y + i));

                double pixel = input[orig_x + orig_y * cols];
                blur += pixel * kernel[(i + half) * kernel_size + (j + half)];
            }

        output[x + y * cols] = static_cast<uchar>(blur);
    }
}

Mat gaussian_blur(const Mat& image, const uint kernel_size) {
    Mat blur(image.rows, image.cols, CV_8UC1);

    // creates pointers for gpu to original image and blur
    uchar *dev_image;
    uchar *dev_blur;

    Mat kernel = getGaussianKernel(kernel_size, 1) * getGaussianKernel(kernel_size, 1).t();
    double *dev_kernel;

    const size_t num_pixels = image.rows * image.cols;

    auto start_mem1 = std::chrono::high_resolution_clock::now();

    // allocate memory on gpu
    hipMalloc((void **)&dev_image, sizeof(uchar) * num_pixels);
    hipMalloc((void **)&dev_blur, sizeof(uchar) * num_pixels);
    hipMalloc((void **)&dev_kernel, sizeof(double) * kernel_size * kernel_size);

    // set blur image memory to 0
    hipMemset(dev_blur, 0, sizeof(uchar) * num_pixels);

    // copy original image and kernel to gpu
    hipMemcpy(dev_image, image.ptr<uchar>(), sizeof(uchar) * num_pixels, hipMemcpyHostToDevice);
    hipMemcpy(dev_kernel, kernel.ptr<double>(), sizeof(double) * kernel_size * kernel_size, hipMemcpyHostToDevice);

    auto stop_mem1 = std::chrono::high_resolution_clock::now();

    const dim3 blockSize(16, 16);
    const dim3 gridSize(image.cols / blockSize.x + 1, image.rows / blockSize.y + 1);

    auto start_kernel = std::chrono::high_resolution_clock::now();

    // calculate blur image
    apply_filter <<< gridSize, blockSize >>> (dev_image, dev_blur, image.rows, image.cols, dev_kernel, kernel_size);

    auto stop_kernel = std::chrono::high_resolution_clock::now();
    auto time_kernel = std::chrono::duration_cast<std::chrono::microseconds>(stop_kernel - start_kernel);
    std::cout << "GPU blur kernel time: " << (double)time_kernel.count() / 1000000 << " sec" << std::endl;

    auto start_mem2 = std::chrono::high_resolution_clock::now();

    // copy result from gpu
    hipMemcpy(blur.data, dev_blur, sizeof(uchar) * num_pixels, hipMemcpyDeviceToHost);

    // free memory on gpu
    hipFree(dev_image); hipFree(dev_blur); hipFree(dev_kernel);

    auto stop_mem2 = std::chrono::high_resolution_clock::now();
    auto time_mem = std::chrono::duration_cast<std::chrono::milliseconds>(stop_mem1 - start_mem1) +
                    std::chrono::duration_cast<std::chrono::milliseconds>(stop_mem2 - start_mem2);
    std::cout << "GPU blur memcpy time: " << (float)time_mem.count() / 1000 << " sec" << std::endl << std::endl;

    return blur;
}